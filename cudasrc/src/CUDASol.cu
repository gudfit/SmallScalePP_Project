#include "hip/hip_runtime.h"
#include "../includes/CUDASol.cuh"
#include "../includes/Transpose.cuh"

#include <hip/hip_runtime.h>

#define TILE_WIDTH 32
#define BLOCK_DIM 32

#define TILE_WIDTH_PADDED (TILE_WIDTH + 1)

__global__ void matmul_kernel(const float *A, const float *B, float *C, int n,
                              int k) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < n && j < n) {
    float sum = 0.0;
    for (int p = 0; p < k; p++) {
      sum += A[i * k + p] * B[p * n + j];
    }
    C[i * n + j] = sum;
  }
}

__global__ void matmul_kernel_shared(const float *A, const float *B, float *C,
                                     int n, int k) {
  __shared__ float tile_A[TILE_WIDTH][TILE_WIDTH];
  __shared__ float tile_B[TILE_WIDTH][TILE_WIDTH];

  int i = blockIdx.y * TILE_WIDTH + threadIdx.y;
  int j = blockIdx.x * TILE_WIDTH + threadIdx.x;

  float sum = 0.0;

  for (int t = 0; t < (k + TILE_WIDTH - 1) / TILE_WIDTH; ++t) {
    int col_A = t * TILE_WIDTH + threadIdx.x;
    if (i < n && col_A < k)
      tile_A[threadIdx.y][threadIdx.x] = A[i * k + col_A];
    else
      tile_A[threadIdx.y][threadIdx.x] = 0.0;

    int row_B = t * TILE_WIDTH + threadIdx.y;
    if (row_B < k && j < n)
      tile_B[threadIdx.y][threadIdx.x] = B[row_B * n + j];
    else
      tile_B[threadIdx.y][threadIdx.x] = 0.0;

    /* Correct synchronization position */
    __syncthreads();

    for (int p = 0; p < TILE_WIDTH; ++p)
      sum += tile_A[threadIdx.y][p] * tile_B[p][threadIdx.x];

    __syncthreads();
  }

  if (i < n && j < n)
    C[i * n + j] = sum;
}
__global__ void matmul_kernel_shared_padded(const float *A, const float *B,
                                            float *C, int n, int k) {
  __shared__ float tile_A[TILE_WIDTH][TILE_WIDTH_PADDED];
  __shared__ float tile_B[TILE_WIDTH_PADDED][TILE_WIDTH];

  int i = blockIdx.y * TILE_WIDTH + threadIdx.y;
  int j = blockIdx.x * TILE_WIDTH + threadIdx.x;

  float sum = 0.0;

  for (int t = 0; t < (k + TILE_WIDTH - 1) / TILE_WIDTH; ++t) {
    int col_A = t * TILE_WIDTH + threadIdx.x;
    if (i < n && col_A < k)
      tile_A[threadIdx.y][threadIdx.x] = A[i * k + col_A];
    else
      tile_A[threadIdx.y][threadIdx.x] = 0.0;

    int row_B = t * TILE_WIDTH + threadIdx.y;
    if (row_B < k && j < n)
      tile_B[threadIdx.y][threadIdx.x] = B[row_B * n + j];
    else
      tile_B[threadIdx.y][threadIdx.x] = 0.0;

    __syncthreads();

    for (int p = 0; p < TILE_WIDTH; ++p)
      sum += tile_A[threadIdx.y][p] * tile_B[p][threadIdx.x];

    __syncthreads();
  }

  if (i < n && j < n)
    C[i * n + j] = sum;
}

void matmul_naive(const float *A, const float *B, float *C, int n, int k) {
  dim3 blockDim(BLOCK_DIM, BLOCK_DIM);
  dim3 gridDim((n + blockDim.x - 1) / blockDim.x,
               (n + blockDim.y - 1) / blockDim.y);

  matmul_kernel<<<gridDim, blockDim>>>(A, B, C, n, k);
  hipDeviceSynchronize();
}

void matmul_shared(const float *A, const float *B, float *C, int n, int k) {
  dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
  dim3 gridDim((n + blockDim.x - 1) / blockDim.x,
               (n + blockDim.y - 1) / blockDim.y);
  matmul_kernel_shared<<<gridDim, blockDim>>>(A, B, C, n, k);
  hipDeviceSynchronize();
}

void matmul_shared_padded(const float *A, const float *B, float *C, int n,
                          int k) {
  dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
  dim3 gridDim((n + blockDim.x - 1) / blockDim.x,
               (n + blockDim.y - 1) / blockDim.y);
  matmul_kernel_shared_padded<<<gridDim, blockDim>>>(A, B, C, n, k);
  hipDeviceSynchronize();
}
// -------------------- TESTING TRANSPOSE ------------------------
__global__ void matmul_kernel_shared_transpose(const float *A, const float *B,
                                               float *C, int n, int k) {
  __shared__ float tile_A[TILE_WIDTH][TILE_WIDTH];
  __shared__ float tile_B[TILE_WIDTH][TILE_WIDTH];

  int i = blockIdx.y * TILE_WIDTH + threadIdx.y;
  int j = blockIdx.x * TILE_WIDTH + threadIdx.x;

  float sum = 0.0;

  for (int t = 0; t < (k + TILE_WIDTH - 1) / TILE_WIDTH; ++t) {
    int col_A = t * TILE_WIDTH + threadIdx.x;
    if (i < n && col_A < k)
      tile_A[threadIdx.y][threadIdx.x] = A[i * k + col_A];
    else
      tile_A[threadIdx.y][threadIdx.x] = 0.0;

    int row_B = t * TILE_WIDTH + threadIdx.y;
    /* Modify B's access to fit the transposed layout */
    if (row_B < k && j < n)
      tile_B[threadIdx.y][threadIdx.x] = B[j * k + row_B];
    else
      tile_B[threadIdx.y][threadIdx.x] = 0.0;

    __syncthreads();

    for (int p = 0; p < TILE_WIDTH; ++p)
      sum += tile_A[threadIdx.y][p] * tile_B[p][threadIdx.x];

    __syncthreads();
  }

  if (i < n && j < n)
    C[i * n + j] = sum;
}

void matmul_shared_BT(const float *A, const float *B, float *C, int n, int k) {
  dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
  dim3 gridDim((n + blockDim.x - 1) / blockDim.x,
               (n + blockDim.y - 1) / blockDim.y);
  matmul_kernel_shared_transpose<<<gridDim, blockDim>>>(A, B, C, n, k);
  hipDeviceSynchronize();
}
/*
 * matmul_ref
 *
 *
 * Reference serial implementation of matrix multiplication.
 * Computes C_ref = A * B, where A is n x k and B is k x n.
 *
 * @param pointer to A,B,C_ref
 * sizes n,k
 * @return void
 */
void matmul_ref(const float *A, const float *B, float *C_ref, int n, int k) {
  /* NOTE: B is stored in j-major order. */
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      float sum = 0.0;
      for (int p = 0; p < k; p++)
        sum += A[i * k + p] * B[p * n + j];
      C_ref[i * n + j] = sum;
    }
  }
}
