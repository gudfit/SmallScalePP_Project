#include "hip/hip_runtime.h"
#include "CUDASol.cuh"
#include <hip/hip_runtime.h>

#define TILE_WIDTH 32
#define BLOCK_DIM 32

#define TILE_WIDTH_PADDED (TILE_WIDTH + 1)

__global__ void matmul_kernel(const double *A, const double *B, double *C,
                              int n, int k) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < n && j < n) {
    double sum = 0.0;
    for (int p = 0; p < k; p++)
      sum += A[i * k + p] * B[p * n + j];

    C[i * n + j] = sum;
  }
}

__global__ void matmul_kernel_shared(const double *A, const double *B,
                                     double *C, int n, int k) {
  __shared__ double tile_A[TILE_WIDTH][TILE_WIDTH];
  __shared__ double tile_B[TILE_WIDTH][TILE_WIDTH];

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  double sum = 0.0;

  for (int t = 0; t < (k + TILE_WIDTH - 1) / TILE_WIDTH; ++t) {

    int col_A = t * TILE_WIDTH + threadIdx.y;
    if (i < n && col_A < k)
      tile_A[threadIdx.x][threadIdx.y] = A[i * k + col_A];
    else
      tile_A[threadIdx.x][threadIdx.y] = 0.0;

    int row_B = t * TILE_WIDTH + threadIdx.x;
    if (j < n && row_B < k)
      tile_B[threadIdx.x][threadIdx.y] = B[row_B * n + j];
    else
      tile_B[threadIdx.x][threadIdx.y] = 0.0;

    __syncthreads();
#pragma unroll
    for (int p = 0; p < TILE_WIDTH; ++p)
      sum += tile_A[threadIdx.x][p] * tile_B[p][threadIdx.y];
    __syncthreads();
  }

  if (i < n && j < n)
    C[i * n + j] = sum;
}

__global__ void matmul_kernel_shared_padded(const double *A, const double *B,
                                            double *C, int n, int k) {
  __shared__ double tile_A[TILE_WIDTH][TILE_WIDTH_PADDED];
  __shared__ double tile_B[TILE_WIDTH_PADDED][TILE_WIDTH];

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  double sum = 0.0;

  for (int t = 0; t < (k + TILE_WIDTH - 1) / TILE_WIDTH; ++t) {

    int col_A = t * TILE_WIDTH + threadIdx.y;
    if (i < n && col_A < k)
      tile_A[threadIdx.x][threadIdx.y] = A[i * k + col_A];
    else
      tile_A[threadIdx.x][threadIdx.y] = 0.0;

    int row_B = t * TILE_WIDTH + threadIdx.x;
    if (j < n && row_B < k)
      tile_B[threadIdx.x][threadIdx.y] = B[row_B * n + j];
    else
      tile_B[threadIdx.x][threadIdx.y] = 0.0;

    __syncthreads();
#pragma unroll
    for (int p = 0; p < TILE_WIDTH; ++p)
      sum += tile_A[threadIdx.x][p] * tile_B[p][threadIdx.y];
    __syncthreads();
  }

  if (i < n && j < n)
    C[i * n + j] = sum;
}

void matmul_naive(const double *A, const double *B, double *C, int n, int k) {
  dim3 blockDim(BLOCK_DIM, BLOCK_DIM);
  dim3 gridDim((n + blockDim.x - 1) / blockDim.x,
               (n + blockDim.y - 1) / blockDim.y);

  matmul_kernel<<<gridDim, blockDim>>>(A, B, C, n, k);
  hipDeviceSynchronize();
}

void matmul_shared(const double *A, const double *B, double *C, int n, int k) {
  dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
  dim3 gridDim((n + blockDim.x - 1) / blockDim.x,
               (n + blockDim.y - 1) / blockDim.y);
  matmul_kernel_shared<<<gridDim, blockDim>>>(A, B, C, n, k);
  hipDeviceSynchronize();
}

void matmul_shared_padded(const double *A, const double *B, double *C, int n,
                          int k) {
  dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
  dim3 gridDim((n + blockDim.x - 1) / blockDim.x,
               (n + blockDim.y - 1) / blockDim.y);
  matmul_kernel_shared_padded<<<gridDim, blockDim>>>(A, B, C, n, k);
  hipDeviceSynchronize();
}

/*
 * matmul_ref
 *
 *
 * Reference serial implementation of matrix multiplication.
 * Computes C_ref = A * B, where A is n x k and B is k x n.
 *
 * @param pointer to A,B,C_ref
 * sizes n,k
 * @return void
 */
void matmul_ref(const double *A, const double *B, double *C_ref, int n, int k) {
  // NOTE: B is stored in j-major order.
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      double sum = 0.0;
      for (int p = 0; p < k; p++)
        sum += A[i * k + p] * B[p * n + j];
      C_ref[i * n + j] = sum;
    }
  }
}
