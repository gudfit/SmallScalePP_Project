#include "hip/hip_runtime.h"
#include "CUDASol.cuh"
#include <hip/hip_runtime.h>

#define TILE_WIDTH 32
#define BLOCK_DIM 32

#define TILE_WIDTH_PADDED (TILE_WIDTH + 1)

__global__ void matmul_kernel(const double *A, const double *B, double *C,
                              int n, int k) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < n && j < n) {
    double sum = 0.0;
    for (int p = 0; p < k; p++)
      sum += A[i * k + p] * B[p * n + j];

    C[i * n + j] = sum;
  }
}

__global__ void matmul_kernel_shared(const double *A, const double *B,
                                     double *C, int n, int k) {
  __shared__ double tile_A[TILE_WIDTH][TILE_WIDTH];
  __shared__ double tile_B[TILE_WIDTH][TILE_WIDTH];

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  double sum = 0.0;

  for (int t = 0; t < (k + TILE_WIDTH - 1) / TILE_WIDTH; ++t) {

    int col_A = t * TILE_WIDTH + threadIdx.y;
    if (i < n && col_A < k)
      tile_A[threadIdx.x][threadIdx.y] = A[i * k + col_A];
    else
      tile_A[threadIdx.x][threadIdx.y] = 0.0;

    int row_B = t * TILE_WIDTH + threadIdx.x;
    if (j < n && row_B < k)
      tile_B[threadIdx.x][threadIdx.y] = B[row_B * n + j];
    else
      tile_B[threadIdx.x][threadIdx.y] = 0.0;

    __syncthreads();
#pragma unroll
    for (int p = 0; p < TILE_WIDTH; ++p)
      sum += tile_A[threadIdx.x][p] * tile_B[p][threadIdx.y];
    __syncthreads();
  }

  if (i < n && j < n)
    C[i * n + j] = sum;
}

__global__ void matmul_kernel_shared_padded(const double *A, const double *B,
                                            double *C, int n, int k) {
  __shared__ double tile_A[TILE_WIDTH][TILE_WIDTH_PADDED];
  __shared__ double tile_B[TILE_WIDTH_PADDED][TILE_WIDTH];

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  double sum = 0.0;

  for (int t = 0; t < (k + TILE_WIDTH - 1) / TILE_WIDTH; ++t) {

    int col_A = t * TILE_WIDTH + threadIdx.y;
    if (i < n && col_A < k)
      tile_A[threadIdx.x][threadIdx.y] = A[i * k + col_A];
    else
      tile_A[threadIdx.x][threadIdx.y] = 0.0;

    int row_B = t * TILE_WIDTH + threadIdx.x;
    if (j < n && row_B < k)
      tile_B[threadIdx.x][threadIdx.y] = B[row_B * n + j];
    else
      tile_B[threadIdx.x][threadIdx.y] = 0.0;

    __syncthreads();
#pragma unroll
    for (int p = 0; p < TILE_WIDTH; ++p)
      sum += tile_A[threadIdx.x][p] * tile_B[p][threadIdx.y];
    __syncthreads();
  }

  if (i < n && j < n)
    C[i * n + j] = sum;
}

void matmul_naive(const double *A, const double *B, double *C, int n, int k) {
  dim3 blockDim(BLOCK_DIM, BLOCK_DIM);
  dim3 gridDim((n + blockDim.x - 1) / blockDim.x,
               (n + blockDim.y - 1) / blockDim.y);

  matmul_kernel<<<gridDim, blockDim>>>(A, B, C, n, k);
  hipDeviceSynchronize();
}

void matmul_shared(const double *A, const double *B, double *C, int n, int k) {
  dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
  dim3 gridDim((n + blockDim.x - 1) / blockDim.x,
               (n + blockDim.y - 1) / blockDim.y);
  matmul_kernel_shared<<<gridDim, blockDim>>>(A, B, C, n, k);
  hipDeviceSynchronize();
}

void matmul_shared_padded(const double *A, const double *B, double *C, int n,
                          int k) {
  dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
  dim3 gridDim((n + blockDim.x - 1) / blockDim.x,
               (n + blockDim.y - 1) / blockDim.y);
  matmul_kernel_shared_padded<<<gridDim, blockDim>>>(A, B, C, n, k);
  hipDeviceSynchronize();
}

/*
 * matmul_ref
 *
 * Reference serial implementation of matrix multiplication with OpenMP
 * parallelization. Computes C_ref = A * B, where A is n x k and B is k x n.
 *
 * @param pointer to A, B, C_ref
 * sizes n, k
 * @return void
 */
void matmul_ref(const double *A, const double *B, double *C_ref, int n, int k) {
/* Initialize C_ref to zeros in parallel */
#pragma omp parallel for
  for (int i = 0; i < n * n; i++)
    C_ref[i] = 0.0;

/* Blocked/tiled implementation with OpenMP parallelism */
#pragma omp parallel for collapse(3)
  for (int ii = 0; ii < n; ii += BLOCK_DIM) {
    for (int jj = 0; jj < n; jj += BLOCK_DIM) {
      for (int kk = 0; kk < k; kk += BLOCK_DIM) {
        for (int i = ii; i < std::min(ii + BLOCK_DIM, n); i++) {
          for (int j = jj; j < std::min(jj + BLOCK_DIM, n); j++) {
            double sum = 0.0;
/* Use SIMD vectorization for the innermost loop */
#pragma omp simd reduction(+ : sum)
            for (int p = kk; p < std::min(kk + BLOCK_DIM, k); p++)
              sum += A[i * k + p] * B[p * n + j];

            C_ref[i * n + j] += sum;
          }
        }
      }
    }
  }
}
