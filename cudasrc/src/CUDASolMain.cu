#include "hip/hip_runtime.h"
#include "CUDASol.cuh"
#include <array>
#include <cmath>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <vector>

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "CUDA error: " << msg << " : " << hipGetErrorString(err)
              << std::endl;
    exit(EXIT_FAILURE);
  }
}

int main() {
  std::vector<int> ns = {512, 1024, 2048, 4096};
  std::vector<int> ks = {32, 48, 64, 96, 128};

  for (int n : ns) {
    for (int k : ks) {
      std::cout << "-------------------------------------------\n";
      std::cout << "Testing: n = " << n << ", k = " << k << "\n";

      /* Pinned Memory (Faster transfer) */
      /* 
      float *A, *B;
      hipHostMalloc(&A, n * k * sizeof(float));
      hipHostMalloc(&B, k * n * sizeof(float)); 
      */

      /* Unity Streams */
      /*
      float *A, *B, *C;
      hipMallocManaged(&A, n * k * sizeof(float), hipMemAttachGlobal);
      hipMallocManaged(&B, k * n * sizeof(float), hipMemAttachGlobal);
      hipMallocManaged(&C, n * n * sizeof(float), hipMemAttachGlobal); 
      */

      float *A = new float[n * k];
      float *B = new float[k * n];
      float *C = new float[n * n];

      std::random_device rd;
      std::mt19937 gen(rd());
      std::uniform_real_distribution<float> dis(0.0f, 1.0f);

      for (int i = 0; i < n * k; i++)
        A[i] = dis(gen);
      for (int i = 0; i < k * n; i++)
        B[i] = dis(gen);

      float *d_A, *d_B, *d_C;
      hipMalloc(&d_A, n * k * sizeof(float));
      hipMalloc(&d_B, k * n * sizeof(float));
      hipMalloc(&d_C, n * n * sizeof(float));
      checkCUDAError("hipMalloc");

      hipMemcpy(d_A, A, n * k * sizeof(float), hipMemcpyHostToDevice);
      hipMemcpy(d_B, B, k * n * sizeof(float), hipMemcpyHostToDevice);
      checkCUDAError("hipMemcpy HostToDevice");

      hipMemset(d_C, 0, n * n * sizeof(float));
      checkCUDAError("hipMemset");

      hipEvent_t start_event, stop_event;
      hipEventCreate(&start_event);
      checkCUDAError("hipEventCreate start");
      hipEventCreate(&stop_event);
      checkCUDAError("hipEventCreate stop");

      hipEventRecord(start_event);
      checkCUDAError("hipEventRecord start");
      matmul_naive(d_A, d_B, d_C, n, k);
      checkCUDAError("matmul_naive");
      hipEventRecord(stop_event);
      checkCUDAError("hipEventRecord stop");

      hipEventSynchronize(stop_event);
      checkCUDAError("hipEventSynchronize");

      float milliseconds = 0;
      hipEventElapsedTime(&milliseconds, start_event, stop_event);
      checkCUDAError("hipEventElapsedTime");
      double mult_time = milliseconds / 1e3;

      hipMemcpy(C, d_C, n * n * sizeof(float), hipMemcpyDeviceToHost);
      checkCUDAError("hipMemcpy DeviceToHost");

      double gflops = (2.0 * k * n * n) / (mult_time * 1e9);
      std::cout << "Multiplication time: " << mult_time << " s\n";
      std::cout << "Performance: " << gflops << " GFLOPS\n";

      float *C_ref = new float[n * n];
      matmul_ref(A, B, C_ref, n, k);

      float max_error = 0.0f;
      for (int i = 0; i < n * n; i++)
        max_error = std::max(max_error, std::fabs(C[i] - C_ref[i]));

      const float tolerance = 1e-6f;
      std::cout << "Max error: " << max_error
                << (max_error < tolerance ? " PASSED" : " FAILED") << "\n";

      /* delete objects and mem */
      
      delete[] A;
      delete[] B;
      delete[] C;

      /* Unified
      hipFree(A);
      hipFree(B);
      hipFree(C);
      */

      /* Pinned Memory
      hipHostFree(A);
      hipHostFree(B);
      */

      delete[] C_ref;

      hipFree(d_A);
      hipFree(d_B);
      hipFree(d_C);
      hipEventDestroy(start_event);
      hipEventDestroy(stop_event);
    }
  }
  return 0;
}
